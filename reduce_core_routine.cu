#include "hip/hip_runtime.h"
// NOT FUNCTIONAL STILL NEEDS TO BE DONE//

template <typename scalar_t>
__global__ void reduce_cuda_kernel(
    const scalar_t* __restrict__ input,
    const scalar_t* __restrict__ reduced_input,
    size_t state_size) {
  const int column = blockIdx.x * blockDim.x + threadIdx.x;
  const int index = blockIdx.y * state_size + column;
  const int gates_row = blockIdx.y * (state_size * 3);
  if (column < state_size) {
    input_gate[index] = sigmoid(gates[gates_row + column]);
    output_gate[index] = sigmoid(gates[gates_row + state_size + column]);
    candidate_cell[index] = elu(gates[gates_row + 2 * state_size + column]);
    new_cell[index] =
        old_cell[index] + candidate_cell[index] * input_gate[index];
    new_h[index] = tanh(new_cell[index]) * output_gate[index];
  }
}

