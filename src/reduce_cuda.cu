#include "hip/hip_runtime.h"
#include <torch/torch.h>

#include "reduce.hh"

template <typename scalar_t>
__global__ void reduce_forward_cuda_kernel(
    scalar_t* __restrict__ reduced,
    const scalar_t* __restrict__ full,
    const int32_t* __restrict__ mapping,
    int32_t n_samples,
    int32_t other_sizes
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n_samples && j < other_sizes) {
       auto reduce_id = mapping[i];
       atomicAdd(reduced + reduce_id * other_sizes + j, full[i * other_sizes + j]);
    //    reduced[reduce_id * other_sizes + j] += full[i * other_sizes + j];
    }
}

void reduce_forward_cuda(
    torch::Tensor& reduced,
    const torch::Tensor& full,
    const torch::Tensor& mapping,
    int n_samples,
    int other_sizes
) {
    CHECK_CUDA(full);
    CHECK_CUDA(reduced);
    CHECK_CUDA(mapping);

    const dim3 threads(1, 64);
    const dim3 blocks(1 + (n_samples - 1) / threads.x, 1 + (other_sizes - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(full.scalar_type(), "reduce_forward_cuda", ([&] {
        reduce_forward_cuda_kernel<<<blocks, threads>>>(
            reduced.data_ptr<scalar_t>(),
            full.data_ptr<scalar_t>(),
            mapping.data_ptr<int32_t>(),
            n_samples,
            other_sizes
        );
    }));
}


template <typename scalar_t>
__global__ void reduce_backward_cuda_kernel(
    scalar_t* __restrict__ full,
    const scalar_t* __restrict__ reduced,
    const int32_t* __restrict__ mapping,
    int32_t n_samples,
    int32_t other_sizes
) {
    // // SOLUTION with for-loop in cuda kernel
    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    // if (i < n_samples) {
    //    auto reduce_id = mapping[i];
    //    for (int j=0; j<other_sizes; j++) {
    //        full[i * other_sizes + j] = reduced[reduce_id * other_sizes + j];
    //    }
    // }

    // SOLUTION without for-loop in cuda kernel
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n_samples && j < other_sizes) {
       auto reduce_id = mapping[i];
       full[i * other_sizes + j] = reduced[reduce_id * other_sizes + j];
    }
}

void reduce_backward_cuda(
    torch::Tensor& full,
    const torch::Tensor& reduced,
    const torch::Tensor& mapping,
    int n_samples,
    int other_sizes
) {
    CHECK_CUDA(full);
    CHECK_CUDA(reduced);
    CHECK_CUDA(mapping);

    // // SOLUTION with for-loop in cuda kernel
    // int threads = 64;
    // int blocks = (n_samples + threads - 1) / threads;

    // SOLUTION without for-loop in cuda kernel
    //const dim3 blocks((n_samples + threads - 1) / threads, (other_sizes + threads - 1) / threads); // tried with one thread but this did not work
    const dim3 threads(1, 64);
    const dim3 blocks(1 + (n_samples - 1) / threads.x, 1 + (other_sizes - 1) / threads.y);

    AT_DISPATCH_FLOATING_TYPES(full.scalar_type(), "reduce_backward_cuda", ([&] {
        reduce_backward_cuda_kernel<<<blocks, threads>>>(
            full.data_ptr<scalar_t>(),
            reduced.data_ptr<scalar_t>(),
            mapping.data_ptr<int32_t>(),
            n_samples,
            other_sizes
        );
    }));
}

void reduce_backward_cudamemcpy(
    torch::Tensor& full,
    const torch::Tensor& reduced,
    const torch::Tensor& mapping,
    int n_samples,
    int other_sizes
) {
    CHECK_CUDA(full);
    CHECK_CUDA(reduced);
    CHECK_CPU(mapping);

    auto to = full.data_ptr<double>();
    auto from = reduced.data_ptr<double>();
    int32_t* mapping_ptr = mapping.data_ptr<int32_t>();

    for (int i = 0; i < n_samples; i++)
    {
        auto to_i = to + other_sizes*i;
        auto reduce_id = mapping_ptr[i];
        auto from_i = from + reduce_id*other_sizes;
        hipMemcpy(to_i, from_i, other_sizes*sizeof(double), hipMemcpyDeviceToDevice);
    }
}
